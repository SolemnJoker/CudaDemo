#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addKernel(float *c, const float *a, const float *b,int nx,int ny)
{
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
    int i = y*nx + x;
	if (y < ny && x < nx)
	{
		c[i] = a[i] + b[i];
	}
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(float* mat_a,float* mat_b,float* mat_c,int nx,int ny)
{
    dim3 block(32, 32);	
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    addKernel<<<grid, block>>>(mat_c,mat_a,mat_b,nx,ny);
}
