#include "hip/hip_runtime.h"
#include "nestedReduce.h"

__global__ void neighboredReduce(int* g_idata,int* g_odata,int size) 
{
    auto tid  = threadIdx.x;
    auto g_idx = threadIdx.x + blockIdx.x*blockDim.x;

    for(unsigned int stride = 1; stride < blockDim.x;stride *= 2)
    {
        if(tid % (2*stride) == 0)
        {
            g_idata[g_idx] += g_idata[g_idx + stride];
        }
        __syncthreads();
    }
    if(tid == 0) g_odata[blockIdx.x] = g_idata[g_idx];
}

void call_neighbored_reduce( int* g_idata,  int* g_odata,dim3 block,dim3 grid)
{
    neighboredReduce<<<grid,block>>>(g_idata,g_odata,block.x);
}

__global__ void gpuRecursiveReduce(int* g_idata, int* g_odata,int size) 
{
    auto tid = threadIdx.x;
    auto g_idx = threadIdx.x + blockDim.x*blockIdx.x;

    if(size == 2 && tid == 0)
    {
        g_odata[tid] = g_idata[0] + g_idata[1]; 
        return ;
    }
    int stride = size / 2;
    if(stride > 1 && tid < stride)
    {
        g_idata[tid] += g_idata[tid + stride];
    }
    __syncthreads();
    if(tid == 0)
    {
        gpuRecursiveReduce<<<1,stride>>>(g_idata + g_idx,g_odata + blockIdx.x,stride);
        hipDeviceSynchronize();
    }
    __syncthreads();
}

void call_recursive_reduce(int* g_idata, int* g_odata,int block,int grid)
{
    gpuRecursiveReduce<<<grid,block>>>(g_idata,g_odata,block.x);
}

