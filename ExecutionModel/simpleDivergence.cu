#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "simpleDivergence.h"


__global__ void mat_kernel1(float * c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = 0;
    float b = 0;
    if(i %2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[i] = a + b;
}

void call_mat_kernel1(float*c ,int size,int block_size){
    dim3 block(block_size,1);
    dim3 grid(size,1);
    mat_kernel1<<<grid,block>>>(c);
}

__global__ void mat_kernel2(float * c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = 0;
    float b = 0;
    if((i/warpSize) %2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[i] = a + b;
}

void call_mat_kernel2(float*c ,int size,int block_size){
    dim3 block(block_size,1);
    dim3 grid(size,1);
    mat_kernel2<<<grid,block>>>(c);
}


__global__ void warmingup(float* c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = 0;
    float b = 0;
    if((i/warpSize) %2 == 0)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[i] = a + b;
}
void call_warmingup(float* c,int size,int block_size){
    dim3 block(block_size,1);
    dim3 grid(size,1);
    warmingup<<<grid,block>>>(c);
}

__global__ void mat_kernel3(float * c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = 0;
    float b = 0;
	int ipred = i % 2;
    if(ipred)
    {
        a = 100.0f;
    }
	if(!ipred)
    {
        b = 200.0f;
    }
    c[i] = a + b;
}

void call_mat_kernel3(float*c ,int size,int block_size){
    dim3 block(block_size,1);
    dim3 grid(size,1);
    mat_kernel3<<<grid,block>>>(c);
}


__global__ void mat_kernel4(float * c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float a = 0;
    float b = 0;
	int ipred = i % 2;
    if(ipred)
    {
        a = 100.0f;
    }
    else
    {
        b = 200.0f;
    }
    c[i] = a + b;
}

void call_mat_kernel4(float*c ,int size,int block_size){
    dim3 block(block_size,1);
    dim3 grid(size,1);
    mat_kernel4<<<grid,block>>>(c);
}









